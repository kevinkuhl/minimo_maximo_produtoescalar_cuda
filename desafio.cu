#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

// Kernel para encontrar o menor valor dado um vetor
__global__ void encontraMenor(int *vetor,int *menor, int tam)
{
        // Calcula a posicao inicial analisada
        int idx = threadIdx.x + blockIdx.x*blockDim.x;
        // Enquanto a posicao for menor do que o limite do vetor
        if (idx < tam)
        {
                // Calcula o elemento minimo, comparando com o valor em menor
                atomicMin(menor,vetor[idx]);
        }
}

// Kernel para encontrar o maior valor dado um vetor
__global__ void encontraMaior(int *vetor,int *maior, int tam)
{
        // Calcula a posicao inicial analisada
        int idx = threadIdx.x + blockIdx.x*blockDim.x;
        // Enquanto a posicao for menor do que o limite do vetor
        if (idx < tam)
        {
                // Calcula o elemento minimo, comparando com o valor em menor
                atomicMax(menor,vetor[idx]);
        }
}

// Kernel para encontrar a soma dos produtos escalares dado um vetor
__global__ void produtoEscalar(int *vetor1, int *vetor2, int *produto, int tam)
{
        // Calcula a posicao inicial analisada
        int idx = threadIdx.x + blockIdx.x*blockDim.x;
        // Enquanto a posicao for menor do que o limite do vetor
        if (idx < tam)
        {
            produto += vetor1[idx] * vetor2[idx];
        }
}

// Funcao para abrir o arquivo e retornar o ponteiro
FILE *abrir_arquivo(char nome[])
{
        FILE *pFile;
        pFile = fopen(nome, "r");
        if(pFile==NULL)
                exit(0);
        return pFile;
}

int main(int argc,char **argv)
{
        if(argc < 2)
            printf("Para utilizar, rode o comando ./calc_matriz <nome da entrada txt>\n");
        // Define as variaveis a serem utilizadas
        FILE *file;
        int *mA_h,*mB_h;
        int *mA_d,*mB_d;

        // Dimensoes das matrizes a serem lidas
        int dimensao;

        // Variaveis de iteracao
        int i,j;

        // Ponteiros para o resultado
        int *menorA_d;
        int *menorA_h;
        int *menorB_d;
        int *menorB_h;
        int *maiorA_d;
        int *maiorA_h;
        int *maiorB_d;
        int *maiorB_h;
        int *produto_h;
        int *produto_d;

        // Definindo as streams
        hipStream_t stream1, stream2, stream3, stream4, stream5;

        // Criando as streams
        hipStreamCreate(&stream1);
        hipStreamCreate(&stream2);
        hipStreamCreate(&stream3);
        hipStreamCreate(&stream4);
        hipStreamCreate(&stream5);

        // Abre o arquivo
        file = abrir_arquivo(argv[1]);

        // Le as dimensoes
        fscanf(file,"%d",&dimensao);

        // Aloca espaco no host para as matrizes e para os resultados
        hipHostMalloc((void**)&mA_h,dimensao*simensao*(sizeof(int)));
        hipHostMalloc((void**)&mB_h,dimensao*simensao*(sizeof(int)));
        hipHostMalloc((void**)&menorA_h,sizeof(int));
        hipHostMalloc((void**)&menorB_h,sizeof(int));
        hipHostMalloc((void**)&maiorA_h,sizeof(int));
        hipHostMalloc((void**)&maiorB_h,sizeof(int));
        hipHostMalloc((void**)&produto_h,sizeof(int));

        // Le as matrizes a partir do arquivo aberto
        for(i=0;i<dimensao;i++)
                for(j=0;j<dimensao;j++)
                        fscanf(file,"%d", &mA_h[i*dimensao+j]);

        for(i=0;i<dimB[0];i++)
                for(j=0;j<dimB[1];j++)
                        fscanf(file,"%d", &mB_h[i*dimensao+j]);
        
        // Fecha o arquivo
        fclose(file);

        // Aloca espaco no device para as matrizes e para os resultados
        hipMalloc((void**)&mA_d,(dimA[0])*dimA[1]*(sizeof(int)));
        hipMalloc((void**)&mB_d,(dimB[0])*dimB[1]* (sizeof(int)));
        hipMalloc((void**)&menorA_d,sizeof(int));
        hipMalloc((void**)&menorB_d,sizeof(int));
        hipMalloc((void**)&maiorA_d,sizeof(int));
        hipMalloc((void**)&maiorB_d,sizeof(int));
        hipMalloc((void**)&produto_d,sizeof(int));


        // Inicializa o conteúdo do resultado no device com 10000
        hipMemset(menorA_d,10000,sizeof(int));
        hipMemset(menorB_d,10000,sizeof(int));
        hipMemset(maiorA_d,-10000,sizeof(int));
        hipMemset(maiorB_d,-10000,sizeof(int));
        hipMemset(produto_d,0,sizeof(int));
        
        // Inicializa as variaveis de thrads por bloco e de blocos por grid
        /* Aqui vale uma ressalva, como as matrizes podem ter dimensoes diferentes
        umas das outras, optamos por utilizar um bloco unico com tamanho total igual
        ao tamanho da matriz (produto das dimensoes).
        Isso evita termos que fazer dois loops for para iterar sobre os blocos (um para cada matriz)
        */
        int threadsPerBlock = dimensao*dimensao;
        int blocksPerGrid = ((dimensao)+threadsPerBlock-1)/threadsPerBlock;

        // Copia asincronamente a memoria do host para o device
        hipMemcpyAsync(mA_d,mA_h,(dimensao*dimensao*sizeof(int)), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(mB_d,mB_h,(dimensao*dimensao*sizeof(int)), hipMemcpyHostToDevice, stream2);
        
        // Chama a funcao para encontrar o minimo na matriz A, utilizando a stream1
        encontraMenor <<<blocksPerGrid,threadsPerBlock,0,stream1>>>(mA_d,menorA_d,dimensao*dimensao);
        // Copia o resultado para o host
        hipMemcpy(menorA_h,menorA_d,sizeof(int), hipMemcpyDeviceToHost);

        encontraMaior <<<blocksPerGrid,threadsPerBlock,0,stream2>>>(mA_d,maiorA_d,dimensao*dimensao);
        hipMemcpy(maiorA_h,maiorA_d,sizeof(int), hipMemcpyDeviceToHost);

        // Chama a funcao para encontrar o minimo na matriz B, utilizando a stream2
        encontraMenor <<<blocksPerGrid,threadsPerBlock,0,stream3>>>(mB_d,menorB_d,dimensao*dimensao);
        // Copia o resultado para o host
        hipMemcpy(menorB_h,menorB_d,sizeof(int), hipMemcpyDeviceToHost);

        encontraMaior <<<blocksPerGrid,threadsPerBlock,0,stream4>>>(mB_d,maiorB_d,dimensao*dimensao);
        hipMemcpy(maiorB_h,maiorB_d,sizeof(int), hipMemcpyDeviceToHost);

        produtoEscalar <<<blocksPerGrid,threadsPerBlock,0,stream5>>>(mA_d,mB_d,produto_d,dimensao*dimensao);
        hipMemcpy(produto_h,produto_d,sizeof(int), hipMemcpyDeviceToHost);

        // Sincroniza as streams criadas
        hipStreamSynchronize(stream1);
        hipStreamSynchronize(stream2);
        hipStreamSynchronize(stream3);
        hipStreamSynchronize(stream4);
        hipStreamSynchronize(stream5);

        // Imprime os resultados
        printf("%d ", *produto_h);
        if(*maiorA_h < *maiorB_h)
            printf("%d ", *maiorB_h);
        else
            printf("%d ", *maiorA_h);
        
        if(*menorA_h < *menorB_h)
            printf("%d\n", *menorA_h);
        else
            printf("%d\n", *menorB_h);


        // Libera o espaco alocado para as variaveis no host
        hipHostFree(menorA_h);
        hipHostFree(menorB_h);
        hipHostFree(maiorA_h);
        hipHostFree(maiorB_h);
        hipHostFree(mA_h);
        hipHostFree(mB_h);
        hipHostFree(produto_h);

        // Libera o espaco alocado para as variaveis no device
        hipFree(mB_d);
        hipFree(mA_d);
        hipFree(menorA_d);
        hipFree(menorB_d);
        hipFree(maiorA_d);
        hipFree(maiorB_d);
        hipFree(produto_d);

        // Libera o espaco alocado para as streams (fecha as streams)
        hipStreamDestroy(stream1);
        hipStreamDestroy(stream2);
        hipStreamDestroy(stream3);
        hipStreamDestroy(stream4);
        hipStreamDestroy(stream5);

        // Sai do programa      
        exit(0);
}